#include "hip/hip_runtime.h"
#include <stdio.h>

#include "cuda_sparse_wrappers.h"

#include <hip/hip_runtime.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>   
#include "cuda_helpers.h"

//implement scan?
extern "C"{
    void prefixSum(unsigned int * csrOffsetsStart,unsigned int csrOffsetsEnd) {
        thrust::inclusive_scan(thrust::device, csrOffsetsStart, csrOffsetsStart + csrOffsetsEnd, csrOffsetsStart); // in-place scan
    }
}

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

__device__ int min_(int l, int r){
    if (r == 0) return l;
    else if(l == 0) return r;
    else{
        if(l < r) return l;
        else return r;
    }
}


__device__ int getGlobalIdx_2D_1D(){
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
    }

//think about minimum
//SHUFFLE UP OR make all threads run through for
__device__ int WarpMin (unsigned int mask,int value, int laneId ){
        for (int offset = 16; offset > 0; offset/=2)
        value = min_(value,__shfl_up_sync(mask, value, offset));
     return value ;
}

__device__ int WarpMax (unsigned int mask,int value){
    
    for (int offset = 8; offset > 0; offset *=2)
    
    value = max(value,__shfl_down_sync(mask, value, offset));
     return value ;
}

__device__ float WarpSum (unsigned int mask,float value, int laneId){
    for (int offset = 16; offset > 0; offset/=2) 
        value += __shfl_down_sync(mask, value, offset);
     return value ;
}


//maybe indexing from 1

__device__ int nextLaneId(int laneId, unsigned int mask){
    int laneIdNum = 1 << (laneId); // laneId == 1 => mask = 000..10
    int laneMask = laneIdNum * 2 - 1;
    int nextLaneId = __ffs(mask & (~laneMask)) - 1;
    return nextLaneId;
}

//C.row[warpId] = ... =>

//make for any to keep threads running, tmp will be the same across threads

// 1 0 2 3      1 0 2 3
// 0 4 0 0   *  0 4 0 0
// 5 0 6 7      5 0 6 7
// 0 8 0 9      0 8 0 9

__global__ void preproccessRows(struct CSR a, struct CSR b, struct CSR c){
    
    #define FULL_MASK 0xffffffff

    int threadId = getGlobalIdx_2D_1D();
    
    int warpId = threadId / 32; //is rowId
    int laneId = threadIdx.x % 32;

    if(warpId < a.M) {

        int aStart = a.row_index[warpId];
        int aEnd = a.row_index[warpId + 1];

        // unsigned int mask = 0;
        unsigned int mask = __ballot_sync(FULL_MASK, (aStart + laneId < aEnd));
        int tmp = 0;
        // int frontCol = 0;
        
        if(aStart + laneId < aEnd){
            int bRowId = a.cols[aStart + laneId];
            
            //elems are adjacent with the mask
            
            for(int rowPos = b.row_index[bRowId]; __any_sync(mask,rowPos < b.row_index[bRowId+1]);tmp++){

                // mask = __ballot_sync(FULL_MASK, (aStart + laneId < aEnd) && (rowPos < b.row_index[bRowId+1]));
                
                int frontCol = rowPos < b.row_index[bRowId+1] ? (b.cols[rowPos] + 1) : 0;

                // printf("WarpId=%i LaneId=%i= frontCol=%i rowPos=%i\n",warpId,laneId,frontCol,rowPos);
                // if (rowPos < b.row_index[bRowId+1]){
                    // frontCol = b.cols[rowPos] + 1;
                // } else {
                    // frontCol = 0;
                // }

                // printf("WarpId=%i, LanedId=%i,Mask=%i, number of 1's=%i\n",warpId,laneId,mask,__popc(mask));
                // int frontCol = b.cols[rowPos] + 1; //+1 to find minimum correctly
                // float frontVal = b.values[rowPos];
                
                //find minimum frontCol ???????
                int frontColMin = WarpMin(mask,frontCol,laneId);
                frontColMin = __shfl_sync(mask,frontColMin,__popc(mask)-1);

                // printf("WarpId=%i, LaneId=%i, frontCol=%i, frontColMin=%i\n",warpId,laneId,frontCol,frontColMin);
                
                if(frontCol == frontColMin) {
                    rowPos++;
                }

            }
            
        }
        if(laneId == 0){
            if (mask == 0){ //empty row
                c.row_index[warpId+1] = 0;    
            } else {
                c.row_index[warpId+1] = tmp;    
            }
            
        }
    }
    
}

__global__ void spGEMMDevice(struct CSR a, struct CSR b, struct CSR c) {
    #define FULL_MASK 0xffffffff
    
    int threadId = getGlobalIdx_2D_1D();
    int warpId = threadId / 32; //is rowId
    int laneId = threadIdx.x % 32;

    if(warpId < a.M) {

        int aStart = a.row_index[warpId];
        int aEnd = a.row_index[warpId + 1];

        unsigned int mask = 0;
        float tmp = 0.0;
        
        if(aStart + laneId < aEnd){
            
            int bRowId = a.cols[aStart + laneId];
            float rowWeight = a.values[aStart + laneId];

            mask = __ballot_sync(FULL_MASK, (aStart + laneId < aEnd));


            int currentCol = 0;
            
            for(int rowPos = b.row_index[bRowId]; __any_sync(mask,rowPos < b.row_index[bRowId+1]); currentCol++){

                // mask = __ballot_sync(FULL_MASK, (aStart + laneId < aEnd) && (rowPos < b.row_index[bRowId+1]));
                
                
                //int frontCol = b.cols[rowPos] + 1; //+1 to find minimum correctly
                //float frontVal = b.values[rowPos];
                int frontCol = rowPos < b.row_index[bRowId+1] ? (b.cols[rowPos] + 1) : 0;
                float frontVal = rowPos < b.row_index[bRowId+1] ? (b.values[rowPos]) : 0.0f;
                //find minimum frontCol
                int frontColMin = WarpMin(mask,frontCol,laneId);
                frontColMin = __shfl_sync(mask,frontColMin,__popc(mask)-1);

                // printf("WarpId=%i, LaneId=%i, frontCol=%i, frontColMin=%i\n",warpId,laneId,frontCol,frontColMin);

                if(frontCol == frontColMin) {
                    rowPos++;
                    tmp = frontVal * rowWeight;
                    // printf("LaneID=%i, a=%f,b=%f,tmp=%f\n",laneId,rowWeight,frontVal,tmp);
                } else {
                    tmp = 0.0;
                }

                float sum = WarpSum(mask,tmp,laneId);
                sum = __shfl_sync(mask,sum,0); //broadcast
                
                if(laneId == 0){
                    // printf("LaneId=%i\n",laneId);
                    // printf("WARP=%i WRITING COL=%i for %i pos, current COL=%i\n",warpId,frontColMin-1,c.row_index[warpId]+currentCol,currentCol);
                    c.values[c.row_index[warpId]+currentCol] = sum;
                    c.cols[c.row_index[warpId]+currentCol] = frontColMin - 1;
                }

            }
            
        }
    }
}

//not more than 65000 rows

int spGEMMCuda(struct CSR* a, struct CSR* b, struct CSR* c){
    
    struct CSR aDevice = {N: a-> N, M: a->M, nnz : a->nnz};
    struct CSR bDevice = {N: b-> N, M: b->M, nnz : b->nnz};
    struct CSR cDevice = {N: b-> N, M : a->M};

    c-> N = b-> N;
    c-> M = a-> M;

    unsigned int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns, *dC_csrOffsets, *dC_columns;
    float *dA_values, *dB_values, *dC_values;
    
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                            (aDevice.M + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, aDevice.nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  aDevice.nnz * sizeof(float)) )
    // allocate B
    CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
                            (bDevice.M + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_columns, bDevice.nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_values,  bDevice.nnz * sizeof(float)) )
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                            (cDevice.M + 1) * sizeof(int)) )

    //copy 
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, a->row_index,
                                (a->M + 1) * sizeof(int),
                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, a->cols, a->nnz * sizeof(int),
                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, a->values,
                                                a->nnz * sizeof(float), hipMemcpyHostToDevice) )
                        // copy B
    CHECK_CUDA( hipMemcpy(dB_csrOffsets, b->row_index,
                                                (b->M + 1) * sizeof(int),
                                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_columns, b->cols, b->nnz * sizeof(int),
                                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_values, b->values,
                                                b->nnz * sizeof(float), hipMemcpyHostToDevice) )

    CHECK_CUDA( hipMemset(dC_csrOffsets,0,(cDevice.M + 1) * sizeof(int)));

    // int block = a->M;

    dim3 grid(1,1,1);

    //
    int k = (a->M / 65535) + 1;
    if (k > 1) {
        grid.x = 65535;
        if(k > 65535){
            grid.y = 65535;
        } else {
            grid.y = k;
        }
        
    } else {
        grid.x = a->M;
    }

    aDevice.cols = dA_columns;
    aDevice.row_index = dA_csrOffsets;
    aDevice.values = dA_values;

    bDevice.cols = dB_columns;
    bDevice.row_index = dB_csrOffsets;
    bDevice.values = dB_values;

    cDevice.row_index = dC_csrOffsets;

    preproccessRows<<<grid,32>>>(aDevice,bDevice,cDevice);
    hipDeviceSynchronize();
    
    // thrust::inclusive_scan(thrust::Tracy Ellisdevice, dC_csrOffsets, dC_csrOffsets + cDevice.M + 1, dC_csrOffsets); // in-place scan
    prefixSum(dC_csrOffsets, cDevice.M + 1);
    // unsigned int crows = 0;
    CHECK_CUDA( hipMemcpy(&(c->nnz), dC_csrOffsets+c->M,
        (1) * sizeof(int), hipMemcpyDeviceToHost) )

    //if zero return empty matrix;

    CHECK_CUDA( hipMalloc((void**) &dC_columns, c->nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  c->nnz * sizeof(float)) )

    cDevice.values = dC_values;
    cDevice.cols = (unsigned int*)dC_columns;
    
    
    spGEMMDevice<<<grid,32>>>(aDevice,bDevice,cDevice);

    hipDeviceSynchronize();

    //allocate C;
    unsigned int* hC_csrOffsets = new unsigned int[c->M + 1];
    unsigned int* hC_columns = new unsigned int[c->nnz];
    float* hC_values = new float[c->nnz];

    c->row_index = hC_csrOffsets;
    c->cols = hC_columns;
    c->values = hC_values;

    CHECK_CUDA( hipMemcpy(c->row_index, dC_csrOffsets, (c->M + 1)* sizeof(int),
                                               hipMemcpyDeviceToHost) )

    CHECK_CUDA( hipMemcpy(c->cols, dC_columns, c->nnz * sizeof(int),
                                               hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(c->values, dC_values,
                                               c->nnz * sizeof(float), hipMemcpyDeviceToHost) )

    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB_csrOffsets) )
    CHECK_CUDA( hipFree(dB_columns) )
    CHECK_CUDA( hipFree(dB_values) )
    CHECK_CUDA( hipFree(dC_csrOffsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )

    return EXIT_SUCCESS;

}



int spGEMMCusparse(struct CSR* a, struct CSR* b, struct CSR* c){
    
    struct CSR aDevice = {N: a-> N, M: a->M, nnz : a->nnz};
    struct CSR bDevice = {N: b-> N, M: b->M, nnz : b->nnz};
    struct CSR cDevice = {N: b-> N, M : a->M};

    c-> N = b-> N;
    c-> M = a-> M;

    unsigned int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns, *dC_csrOffsets, *dC_columns;
    float *dA_values, *dB_values, *dC_values;

    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE; //only this is supported
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE; //
    hipDataType        computeType = HIP_R_32F; //32-bit IEEE-754 real float
    
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                            (aDevice.M + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, aDevice.nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  aDevice.nnz * sizeof(float)) )
    // allocate B
    CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
                            (bDevice.M + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_columns, bDevice.nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_values,  bDevice.nnz * sizeof(float)) )
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                            (cDevice.M + 1) * sizeof(int)) )

    //copy 
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, a->row_index,
                                (a->M + 1) * sizeof(int),
                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, a->cols, a->nnz * sizeof(int),
                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, a->values,
                                                a->nnz * sizeof(float), hipMemcpyHostToDevice) )
                        // copy B
    CHECK_CUDA( hipMemcpy(dB_csrOffsets, b->row_index,
                                                (b->M + 1) * sizeof(int),
                                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_columns, b->cols, b->nnz * sizeof(int),
                                                hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_values, b->values,
                                                b->nnz * sizeof(float), hipMemcpyHostToDevice) )

    CHECK_CUDA( hipMemset(dC_csrOffsets,0,(cDevice.M + 1) * sizeof(int)));

    aDevice.cols = dA_columns;
    aDevice.row_index = dA_csrOffsets;
    aDevice.values = dA_values;

    bDevice.cols = dB_columns;
    bDevice.row_index = dB_csrOffsets;
    bDevice.values = dB_values;

    cDevice.row_index = dC_csrOffsets;

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, aDevice.M, aDevice.N, aDevice.nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, bDevice.M, bDevice.N, bDevice.nnz,
                                      dB_csrOffsets, dB_columns, dB_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, aDevice.M, bDevice.N, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    //--------------------------------------------------------------------------

    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requiremnent for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2) )
    // get matrix C non-zero entries C_num_nnz1
    int64_t C_num_rows1, C_num_cols1, C_num_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
                                         &C_num_nnz1) )
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_num_nnz1 * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_num_nnz1 * sizeof(float)) )
    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values) )

    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )

    // destroy matrix/vector descriptors

    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    

    
    cDevice.values = dC_values;
    cDevice.cols = (unsigned int*)dC_columns;

    c->nnz = C_num_nnz1;
    
    
    
    //allocate C;
    unsigned int* hC_csrOffsets = new unsigned int[c->M + 1];
    unsigned int* hC_columns = new unsigned int[c->nnz];
    float* hC_values = new float[c->nnz];

    c->row_index = hC_csrOffsets;
    c->cols = hC_columns;
    c->values = hC_values;

    CHECK_CUDA( hipMemcpy(c->row_index, dC_csrOffsets, (c->M + 1)* sizeof(int),
                                               hipMemcpyDeviceToHost) )

    CHECK_CUDA( hipMemcpy(c->cols, dC_columns, c->nnz * sizeof(int),
                                               hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(c->values, dC_values,
                                               c->nnz * sizeof(float), hipMemcpyDeviceToHost) )

    
    CHECK_CUDA( hipFree(dBuffer1) )
    CHECK_CUDA( hipFree(dBuffer2) )                                            
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB_csrOffsets) )
    CHECK_CUDA( hipFree(dB_columns) )
    CHECK_CUDA( hipFree(dB_values) )
    CHECK_CUDA( hipFree(dC_csrOffsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )

    return EXIT_SUCCESS;

}
//find minimum
//find sum

// int main(int argc, char** argv){
    
//     const unsigned int A_num_rows = 4;
//     const unsigned int A_num_cols = 4;
//     const unsigned int A_num_nnz  = 9;
//     const unsigned int B_num_rows = 4;
//     const unsigned int B_num_cols = 4;
//     const unsigned int B_num_nnz  = 8;
//     unsigned int   hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
//     unsigned int   hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
//     float hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
//                               6.0f, 7.0f, 8.0f, 9.0f };
//     // int   hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
//     // int   hA_csrOffsets[] = { 0, 3, 3, 7, 8 };
//     // int   hA_columns[]    = { 0, 2, 3, 2, 0, 2, 3, 1, 3 };
//     // float hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
//                             //   6.0f, 7.0f, 8.0f, 9.0f };
//     unsigned int   hB_csrOffsets[] = { 0, 2, 4, 7, 8 };
//     unsigned int   hB_columns[]    = { 0, 3, 1, 3, 0, 1, 2, 1 };
//     float hB_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
//                               6.0f, 7.0f, 8.0f };
//     unsigned int   hC_csrOffsetsTest[] = { 0, 4, 6, 10, 12 };
//     // int hC_csrOffsetsTest[5] = {0};
//     unsigned int   hC_columns[]    = { 0, 1, 2, 3, 1, 3, 0, 1, 2, 3, 1, 3 };
//     float hC_values[]     = { 11.0f, 36.0f, 14.0f, 2.0f,  12.0f,
//                               16.0f, 35.0f, 92.0f, 42.0f, 10.0f,
//                               96.0f, 32.0f };

//     struct CSR csrA = {N : A_num_cols, M : A_num_rows, nnz : A_num_nnz, values : hA_values, cols : hA_columns, row_index : hA_csrOffsets };
//     struct CSR csrB = {N : B_num_cols, M : B_num_rows, nnz : B_num_nnz, values : hB_values, cols : hB_columns, row_index : hB_csrOffsets };
//     struct CSR csrC = {};

//     spGEMMCuda(&csrA,&csrB,&csrC);


//     for(int i = 0; i < csrC.nnz; i++) {
//         printf("%f ",csrC.values[i]);
//     }
//     printf("\n");

//     delete[] csrC.values;
//     delete[] csrC.cols;
//     delete[] csrC.row_index;

    
//     // int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns, *dC_csrOffsets, *dC_columns;
//     // float *dA_values, *dB_values, *dC_values;
//     // // allocate A
//     // CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
//     //                         (A_num_rows + 1) * sizeof(int)) )
//     // CHECK_CUDA( hipMalloc((void**) &dA_columns, A_num_nnz * sizeof(int))   )
//     // CHECK_CUDA( hipMalloc((void**) &dA_values,  A_num_nnz * sizeof(float)) )
//     // // allocate B
//     // CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
//     //                         (B_num_rows + 1) * sizeof(int)) )
//     // CHECK_CUDA( hipMalloc((void**) &dB_columns, B_num_nnz * sizeof(int))   )
//     // CHECK_CUDA( hipMalloc((void**) &dB_values,  B_num_nnz * sizeof(float)) )
//     // // allocate C offsets
//     // CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
//     //                         (A_num_rows + 1) * sizeof(int)) )
                    
// // copy A
//     // CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
//     //                     (A_num_rows + 1) * sizeof(int),
//     //                     hipMemcpyHostToDevice) )
//     // CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_num_nnz * sizeof(int),
//     //                     hipMemcpyHostToDevice) )
//     // CHECK_CUDA( hipMemcpy(dA_values, hA_values,
//     //                                            A_num_nnz * sizeof(float), hipMemcpyHostToDevice) )
//     //                     // copy B
//     // CHECK_CUDA( hipMemcpy(dB_csrOffsets, hB_csrOffsets,
//     //                                            (B_num_rows + 1) * sizeof(int),
//     //                                            hipMemcpyHostToDevice) )
//     // CHECK_CUDA( hipMemcpy(dB_columns, hB_columns, B_num_nnz * sizeof(int),
//     //                                            hipMemcpyHostToDevice) )
//     // CHECK_CUDA( hipMemcpy(dB_values, hB_values,
//     //                                            B_num_nnz * sizeof(float), hipMemcpyHostToDevice) )

//     // CHECK_CUDA( hipMemset(dC_csrOffsets,0,(A_num_rows + 1) * sizeof(int)));
    
//     // struct CSR csrA = CSR{N : A_num_cols, M : A_num_rows, nnz : A_num_nnz, values : dA_values, cols : (unsigned int*)dA_columns, row_index : (unsigned int*)dA_csrOffsets};
//     // struct CSR csrB = CSR{N : B_num_cols, M : B_num_rows, nnz : B_num_nnz, values : dB_values, cols : (unsigned int*)dB_columns, row_index : (unsigned int*)dB_csrOffsets};
//     // struct CSR csrC = CSR{N : B_num_cols, M : A_num_rows, nnz : 0, values : NULL, cols : NULL, row_index : (unsigned int*)dC_csrOffsets};

//     // preproccessRows<<<4,32>>>(csrA,csrB,csrC);
//     // hipDeviceSynchronize();
    
//     // thrust::inclusive_scan(thrust::device, dC_csrOffsets, dC_csrOffsets + A_num_rows + 1, dC_csrOffsets); // in-place scan

//     // CHECK_CUDA( hipMemcpy(hC_csrOffsetsTest, dC_csrOffsets,
//     //     (A_num_rows + 1) * sizeof(int), hipMemcpyDeviceToHost) )

//     // // allocate C
//     // csrC.nnz = hC_csrOffsetsTest[A_num_rows];
//     // printf("C's nnz=%i\n",csrC.nnz);
//     // CHECK_CUDA( hipMalloc((void**) &dC_columns, csrC.nnz * sizeof(int))   )
//     // CHECK_CUDA( hipMalloc((void**) &dC_values,  csrC.nnz * sizeof(float)) )

//     // csrC.values = dC_values;
//     // csrC.cols = (unsigned int*)dC_columns;
    
//     // for(int i = 0; i < A_num_rows + 1; i++) {
//     //     printf("%i ",hC_csrOffsetsTest[i]);
//     // }
//     // printf("\n");

//     // spGEMMDevice<<<1,128>>>(csrA,csrB,csrC);

//     // CHECK_CUDA( hipMemcpy(hC_columns, dC_columns, csrC.nnz * sizeof(int),
//     //                                            hipMemcpyDeviceToHost) )
//     // CHECK_CUDA( hipMemcpy(hC_values, dC_values,
//     //                                            csrC.nnz * sizeof(float), hipMemcpyDeviceToHost) )

//     // for(int i = 0; i < csrC.nnz; i++) {
//     //     printf("%f ",hC_values[i]);
//     // }
//     // printf("\n");

//     // CHECK_CUDA( hipFree(dA_csrOffsets) )
//     // CHECK_CUDA( hipFree(dA_columns) )
//     // CHECK_CUDA( hipFree(dA_values) )
//     // CHECK_CUDA( hipFree(dB_csrOffsets) )
//     // CHECK_CUDA( hipFree(dB_columns) )
//     // CHECK_CUDA( hipFree(dB_values) )
//     // CHECK_CUDA( hipFree(dC_csrOffsets) )
//     // CHECK_CUDA( hipFree(dC_columns) )
//     // CHECK_CUDA( hipFree(dC_values) )

//     // copy C back

//     //todo make wrapper and write tests

// }